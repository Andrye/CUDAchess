#include "hip/hip_runtime.h"
#include "chess.h"
#include <cstdint>
#include <cstdio>
#include <vector>

void launchKernel(){
    const int nThreads = 1024;
    Game* games = (Game*) malloc(sizeof(Game) * nThreads);
    for(int i = 0; i < nThreads; i++){
        games[i] = Game();
    }

    Game *devGames;
    Move *devMoves;
    float* devValues;

    hipMalloc((void**) &devMoves, sizeof(Move) * nThreads);
    hipMalloc((void**) &devValues, sizeof(float) * nThreads);
    hipMalloc((void**) &devGames, sizeof(Game) * nThreads);
    hipMemcpy(devGames, games, sizeof(Game) * nThreads, hipMemcpyHostToDevice);
    dim3 numThreads(nThreads, 1, 1);

    alphaBeta<<<1, numThreads>>>(devGames, devMoves, devValues);
    hipDeviceSynchronize();
}



int main(){

    launchKernel();
    printf("Implement me\n");
    return 0;
}
