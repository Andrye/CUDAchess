#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <cstdio>
#include <iostream>
#include "node.h"

extern const float INF = 1600000000;
extern const float NODE_INACCESSIBLE = -INF;

__host__
std::ostream &operator <<(std::ostream &os, node const &n){
    for(int x = 0; x < 4; x++) os << "x" << x << ":                  "; os << "\n";
    for(int x = 0; x < 4; x++) os <<"    y0  y1  y2  y3   "; os << "\n";
    char const* h = "  +---+---+---+---+  ";
    os << h << h << h << h << "\n";
    for(int z = 0; z < 4; z++){
        for(int x = 0; x < 4; x++){
            os << "z" << z;
            for(int y = 0; y < 4; y++){
                int bit = x + 4 * y + 16 * z;
                os << "| " << ((n.xs>>bit &1)?'X':((n.os>>bit)&1)?'O':' ') << " ";
            }
            os << "|  ";
        }
        os << "\n" << h << h << h << h << "\n";
    }
    return os;
}

__host__
inline int get_code_hash(std::string const &line)
{
	return line[0] - '0' + 4 * (line[2] - '0') + 16 * (line[4] - '0');
}

__host__
int parse_move(std::string const &line){
    if (line[0] < '0' || '3' < line[0] || line[1] != ' ' || 
        line[2] < '0' || '3' < line[2] || line[3] != ' ' ||
        line[4] < '0' || '3' < line[4] || (line[5] != '\0' && line[5] != '\n'))
        return -1;
    return get_code_hash(line);
}

unsigned int get_console_move(node const &current_node){
    std::cout << current_node << "node value:" << value(current_node) << std::endl;
    int move = -1;
    do {
        std::cout << "your move" << std::endl;
        std::string line;
        std::getline(std::cin, line);
        move = parse_move(line);
    } while(move < 0 || !get_child(current_node, move, nullptr));
    return (unsigned int) move;
}




extern const int n_children = 64;

__host__
void init_node(node *n){
    n->xs = n->os = 0;
}

//Thank you, wikipedia!
const uint64_t m1  = 0x5555555555555555;
const uint64_t m2  = 0x3333333333333333;
const uint64_t m4  = 0x0f0f0f0f0f0f0f0f;
//const uint64_t m8  = 0x00ff00ff00ff00ff;
//const uint64_t m16 = 0x0000ffff0000ffff;
//const uint64_t m32 = 0x00000000ffffffff;
//const uint64_t hff = 0xffffffffffffffff;
const uint64_t h01 = 0x0101010101010101;

__host__ __device__
int popcount(uint64_t x){
    x -= (x >> 1)& m1;
    x = (x & m2) + ((x >> 2) & m2);
    x = (x + (x >> 4)) & m4;
    return (x * h01) >> 56;
}

__host__ __device__
bool get_child(node const& parent, unsigned int id, node *d_child){
    if(((parent.xs | parent.os) >> id) & 1){
        return false;
    }
    if(d_child != nullptr){
        d_child->os = parent.xs;
        d_child->xs = parent.os + (((uint64_t)1) << id);
    }
    return true;
}

__host__ __device__
int count_line(node const *n, uint64_t line){
    int count = 0;
    if((n->xs & line) == (uint64_t) 0) count++;
    if((n->os & line) == (uint64_t) 0) count--;
    return count;
}

uint64_t make_line(int x0, int dx, int y0, int dy, int z0, int dz){
    uint64_t line = 0;
    for(int i = 0; i < 4; i++){
        line = line | (((uint64_t) 1) << (x0 + i*dx) + 4*(y0 + i*dy) + 16*(z0 + i*dz));
    }
    return line;

}

__host__ __device__
int line_type(node const &n, uint64_t line){
    int os = popcount(n.os & line);
    int xs = popcount(n.xs & line);
    if((os == 0) == (xs == 0)) return 0;
    return os ? os : xs + 4;
}
// stats[0] := number of lines with no Os or Xs or with both Os and Xs  //Os = mine, Xs = yours
// stats[i=1..4] := number of lines with i Os and no Xs
// stats[i=5..8] := number of lines with i-4 Xs and no Os;
__host__ __device__
void line_stats(node const &n, int* stats){
    //"1D" lines:
    for(int i = 0; i < 4; i++)
        for(int j = 0; j < 4; j++){
            stats[line_type(n, 0x000000000000000full << (4 * i + 16 * j))]++;
            stats[line_type(n, 0x0000000000001111ull << (i + 16 * j))]++;
            stats[line_type(n, 0x0001000100010001ull << (i + 4 * j))]++;
        }

    //"2D" lines:
    for(int i = 0; i < 4; i++){
        stats[line_type(n, 0x1000010000100001ull << i)]++;
        stats[line_type(n, 0x0001001001001000ull << i)]++;
        stats[line_type(n, 0x0008000400020001ull << (4 * i))]++;
        stats[line_type(n, 0x0001000200040008ull << (4 * i))]++;
        stats[line_type(n, 0x0000000000008421ull << (16 * i))]++;
        stats[line_type(n, 0x0000000000001248ull << (16 * i))]++;
    }

    //"3D" lines:
    stats[line_type(n, 0x0001002004008000ull)]++;
    stats[line_type(n, 0x0008004002001000ull)]++;
    stats[line_type(n, 0x1000020000400008ull)]++;
    stats[line_type(n, 0x8000040000200001ull)]++;
}


__host__ __device__
float value(node const& n){
    int stats[9] = {};
    line_stats(n, stats);
    if(stats[4]) return INF;
    if(stats[8]) return -INF;
    float c = 1.;
    float v = 0;
    for(int i = 1; i <= 4; i++){
        v += c * (stats[i] - stats[i + 4]);
        c *= 152;
    }
    return v;
}

__host__
bool is_terminal(node const &n){
    assert((n.xs & n.os) == 0);
    if(popcount(n.xs | n.os) == 64) return true;
    int stats[9] = {};
    line_stats(n, stats);
    return stats[4] || stats[8];
}
