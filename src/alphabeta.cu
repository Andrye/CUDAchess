#include "hip/hip_runtime.h"
#include "alphabeta.h"
#include <stdio.h>

//extern const float INF, NODE_INACCESSIBLE;

const float INF = 1600000000;
const float NODE_INACCESSIBLE = -INF;
//TODO: how do we make extern const variables visible on device apart from stupid copying?
//If there is no elegant solution, then macro?

struct AB
{
	float a, b;
    
    __host__ __device__
	AB(){}
    
    __host__ __device__
	AB(float a, float b) : a(a), b(b) {}
};


__global__
void compute_children_of_a_node (node *nodes, float *dev_values, node * current_node, unsigned int depth, int n_children, AB limit);

__device__
float compute_node(node const &current_node, unsigned int depth, int n_children, AB limit);

__host__
int get_best_index(float * d_values, int n_children);


__host__ __device__
float invert(float limit)
{
	return -limit;
}
__host__ __device__
AB invert (AB val)
{
	return AB( invert(val.b), invert(val.a) );
}


/* nodes - for now it's used as a free space for compute_other_nodes(). Maybe it'll change later.
 * depth - not fully implemented as of yet
 * best_move - can be nullptr if we only want the numerical result
 */
__host__
float alpha_beta(node * nodes, float * d_values, node const &current_node, unsigned int depth, int n_children, unsigned int * best_move_value, dim3 numThreads) //TODO: for now it's assumed n_children < legth of "nodes" array
{
    if(depth == 0 || is_terminal(current_node))
	{
		return invert(value(current_node));
	}


	node child;
	
	int __index_of_recursive_estimation; //this variable can probably be deleted in the final version, but is crucial untill GPU has recursion as well as CPU
	for (int i=0; i<n_children; i++)
        if(get_child(current_node, i, &child))
	    {
	        __index_of_recursive_estimation = i;
	        break;//TODO: not 0, but what? random? some heura to find a sound candidate?
	    }

	float limit_estimation = alpha_beta(nodes, d_values, child, depth - 1, n_children, nullptr, numThreads);
 //   pnrintf("estim %f\n", limit_estimation);//, uint3(numThreads));	
    float * values = new float[n_children]; //it'll be done better in the future,
    
    node * dev_current_node;
    hipMalloc((void**) &dev_current_node, sizeof(node));
    hipError_t cudaResult = hipMemcpy(dev_current_node, &current_node, sizeof(node), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        printf("cuda error  %d\n", cudaResult);
        throw 1;
    }

	compute_children_of_a_node <<<1, numThreads>>> (nodes, d_values, dev_current_node, depth, n_children, AB(limit_estimation, INF));
    cudaResult = hipDeviceSynchronize();
    if(cudaResult != hipSuccess)
    {
        printf("cuda error %s\n", hipGetErrorString(cudaResult));
        throw 1;
    }
    
        //i.e. taking the best will be on GPU
    cudaResult = hipMemcpy(values, d_values, sizeof(float) * n_children, hipMemcpyDeviceToHost);
    if(cudaResult != hipSuccess)
    {
        printf("cuda error  %d\n", cudaResult);
        throw 1;
    }
	int best_ind = get_best_index(values, n_children);
	float result = values[best_ind];
	
    /******** can be deleted in the final version ********/
    if(result <= limit_estimation)
        best_ind = __index_of_recursive_estimation;
            
	/*if(best_move != nullptr)
    {a
        //don't look at this code, it's stupid. But I want to finish it now
		hipMemcpy(best_move, nodes + best_ind, sizeof(node), hipMemcpyDeviceToHost);
	//    printf("  %d\n", (int)(best_move->xs + best_move->os));
    }
    */
	delete[] values;
	//hipFree(dev_current_node);
	if(best_move_value != nullptr)
	    *best_move_value = best_ind;
	
//	printf("res %f\n", result);
	return invert(result);

	//float best_val = thrust::reduce(d_values, d_values + n_children, thrust::maximum<float>); //TODO: can we use library magic or should we paste out code for scan?    
   
}

__global__
void compute_children_of_a_node (node *nodes, float *values, node * current_node, unsigned int depth, int n_children, AB limit)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    //int bl_id = blockIdx.x; //TODO: no blocks so far
    //int th_id = threadIdx.x;
  //  printf("%d\n", id);
    node child;
    node * childptr = &child;
    //printf("%d: %u\n", id, *current_node);
    
    if(get_child(*current_node, id, childptr))
        values[id] = compute_node(child, depth - 1, n_children, invert(limit));
    else
        values[id] =-NODE_INACCESSIBLE;
    values[id] = invert(values[id]);
}

__device__
float compute_node(node const &current_node, unsigned int depth, int n_children, AB limit)
{
	if(is_terminal(current_node))
		return value(current_node);
	
	node child;
	float best_res = -INF;
	
	for (int i=0; i<n_children; i++)
	{
		if(!get_child(current_node, i, &child))
		    continue;
		float temp_res = invert(value(child)); //recursion here should be
		
		if(temp_res > best_res)
		{
			best_res = temp_res;
			if(temp_res > limit.a)
			{
				if(limit.a >= limit.b)
					return INF; 				//alpha-beta prunning
			}
		}
	}
	return best_res;
}
		
    
//maybe we'll paste code from C instead of this function, but will it be any imporvement at all?
__host__
int get_best_index(float * values, int n_children)
{
	int res_index;
	float val = -1e10;
	for (int i=0; i<n_children; i++)
	{
		if(values[i] > val)
		{
			val = values[i];
			res_index = i;
		}
	}
	return res_index;
}

