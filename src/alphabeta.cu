#include "hip/hip_runtime.h"
#include "alphabeta.h"
#include <algorithm>
#include <stdio.h>


/********** Note that, unlike theearly versions, every funtion returns the value of node X
            for the player who is on the move in node X. It it a calling function's responsibility to invert
            the value. This should only be changed consistently everywhere in this file.
***********/

//extern const float INF, NODE_INACCESSIBLE;

const float INF = 1600000000;
const float NODE_INACCESSIBLE = -INF;
//TODO: how do we make extern const variables visible on device apart from stupid copying?
//If there is no elegant solution, then macro?

struct AB
{
	float a, b;
    
    __host__ __device__
	AB(){}
    
    __host__ __device__
	AB(float a, float b) : a(a), b(b) {}
};

struct stack_entry {

    AB limits;
    node current_node;
    int color;
    int idx; // id of child to be searched.
    char valid_children[(N_CHILDREN + 7) / 8]; // bit mask of valid children

};

__global__
void compute_children_of_a_node (node *nodes, float *dev_values, node * current_node, unsigned int depth, AB limit);

__device__
float compute_node(node const &current_node, unsigned int depth, AB limit);

__host__
int get_best_index(float * d_values);


__host__ __device__
float invert(float limit)
{   
	return -limit;
}
__host__ __device__
AB invert (AB val)
{
	return AB( invert(val.b), invert(val.a) );
}


/* nodes - unused till we implementbclocks
 * best_move - can be nullptr if we only want the numerical result
 */
__host__
float alpha_beta(node * nodes, float * d_values, node const &current_node, unsigned int depth, unsigned int * best_move_value, dim3 numThreads) //TODO: for now it's assumed N_CHILDREN < legth of "nodes" array
{
    if(depth == 0 || is_terminal(current_node))
	{
		return value(current_node);
	}


	node child;
	
	int __index_of_recursive_estimation; //this variable can probably be deleted in the final version, but is crucial untill GPU has recursion as well as CPU
	for (int i=0; i<N_CHILDREN; i++)     //it should be sort, shouldn't it?
        if(get_child(current_node, i, &child))
	    {
	        __index_of_recursive_estimation = i;
	        break;
	    }

	float limit_estimation = invert( alpha_beta(nodes, d_values, child, depth - 1, nullptr, numThreads) );
    
    float * values = new float[N_CHILDREN]; //it'll be done better in the future,
    
    node * dev_current_node;
    hipMalloc((void**) &dev_current_node, sizeof(node));
    hipError_t cudaResult = hipMemcpy(dev_current_node, &current_node, sizeof(node), hipMemcpyHostToDevice);
    if(cudaResult != hipSuccess)
    {
        printf("cuda error  %d\n", cudaResult);
        throw 1;
    }

	compute_children_of_a_node <<<1, numThreads>>> (nodes, d_values, dev_current_node, depth, AB(limit_estimation, INF));
    cudaResult = hipDeviceSynchronize();
    if(cudaResult != hipSuccess)
    {
        printf("cuda error %s\n", hipGetErrorString(cudaResult));
        throw 1;
    }
    
    cudaResult = hipMemcpy(values, d_values, sizeof(float) * N_CHILDREN, hipMemcpyDeviceToHost);
    if(cudaResult != hipSuccess)
    {
        printf("cuda error  %d\n", cudaResult);
        throw 1;
    }
	int best_ind = get_best_index(values);
	float result = values[best_ind];
	
    /******** can be deleted in the final version ********/
    if(result <= limit_estimation)
        best_ind = __index_of_recursive_estimation;
            
	/*if(best_move != nullptr)
    {
        //don't look at this code, it's stupid. But I want to finish it now
		hipMemcpy(best_move, nodes + best_ind, sizeof(node), hipMemcpyDeviceToHost);
    }*/

	delete[] values;
	hipFree(dev_current_node);
	if(best_move_value != nullptr)
	    *best_move_value = best_ind;
	
	return result;

	//float best_val = thrust::reduce(d_values, d_values + N_CHILDREN, thrust::maximum<float>); //TODO: can we use library magic or should we paste out code for scan?    
   
}

const int MAX_STACK_SIZE = 10;
__global__ 
void alpha_beta_gpu(node *nodes, float *values, unsigned int depth, AB limits){

    __shared__ stack_entry stack[MAX_STACK_SIZE];
    __shared__ stack_entry* stacklast;
    __shared__ char valid_children[N_CHILDREN];
    __shared__ bool toContinue;
    __shared__ float children_values[N_CHILDREN];

    int thid = threadIdx.x;
    int blid = blockIdx.x;
    node local_node;
    float ret;


    
    if(thid == 0){

        stack[0].limits = limits;
        stack[0].current_node = nodes[blid];
        stack[0].color = 1;
        stack[0].idx = 0;
        stacklast = stack;
    }
    __syncthreads();
    while(stacklast >= stack){
        if(thid == 0){
            toContinue = false;
        }

        if(thid == 0 && is_terminal(stacklast->current_node)){ // if current node is terminal
            float val = stacklast->color * value(stacklast->current_node);
            ret = val;
            toContinue = true;
        } else if(stacklast == stack + depth){ // if max depth reached
            
            if(get_child(stacklast->current_node, thid, &local_node)){ // find values of children
                children_values[thid] = value(local_node);
            } else {
                children_values[thid] = INF;
            }
            
            for(int d = 1; d < N_CHILDREN; d <<= 1){
                __syncthreads();
                if((thid & d) == 0 && (thid | d) < N_CHILDREN){ // find min of these values
                    float val = children_values[thid | d];
                    if (val < children_values[thid])
                        children_values[thid] = val;
                }
            }

            if(thid == 0)
                ret = children_values[0];

            toContinue = true;
        } else { // we've just returned from recursion.
            if(-ret > stacklast->limits.a){
                stacklast->limits.a = -ret;
            }
            if(stacklast->limits.a >= stacklast->limits.b){
                stacklast->idx = N_CHILDREN;
            }
        }

        __syncthreads();

        if(toContinue){
            continue;
        }
        
        if(stacklast->idx == 0){ // if this is the first time we are at current node
            bool has_child = get_child(stacklast->current_node, thid, nullptr) ? (thid & 0xf) : 0;
            // we want to calculate bit mask of valid children. First we will find singleton masks
            // and then we will run (bitwise or)-scan.
            valid_children[thid] = has_child ? 1 << (thid & 7) : 0;

            //note that no syncthreading is needed below - communication is within warp;
            for(int d = 1; d < 8; d <<= 1){
                if((thid & d) == 0 && (thid | d) < N_CHILDREN)
                    valid_children[thid] = valid_children[thid] | valid_children[thid | d];
            }
            if(thid & 7 == 0)
                stacklast->valid_children[thid >> 3] = valid_children[thid];
        }
        __syncthreads();
        if(thid == 0){
            int idx = stacklast->idx;
            //find valid idx
            while(idx < N_CHILDREN && ((stacklast->valid_children[idx >> 3] >> (idx & 7)) & 1) == 0)
                idx++;

            if(idx == N_CHILDREN){ // if all children searched - return from recursion
                ret = limits.a;
                stacklast--;
            } else { // otherwise search children.

                (stacklast+1)->limits.a = -stacklast->limits.b;
                (stacklast+1)->limits.b = -stacklast->limits.a;
                get_child(stacklast->current_node, idx, &(stacklast+1)->current_node);
                (stacklast+1)->color = -stacklast->color;
                (stacklast+1)->idx = 0;

                stacklast->idx = ++idx;
                stacklast++;
            }
        }
        __syncthreads();


    }

    values[blid] = ret;
}

unsigned int get_alpha_beta_gpu_move(node const &n){
    
    const int depth = 3;
    bool is_node[N_CHILDREN];
    node nodes[N_CHILDREN];
    int children_cnt = 0;
    
    for(int i = 0; i < N_CHILDREN; i++){
        if(is_node[i] = get_child(n, i, &nodes[children_cnt]))
            children_cnt++;
    }
    
    node* dev_nodes;
    float* dev_values;
    hipMalloc((void**) &dev_nodes, sizeof(node) * children_cnt);
    hipMalloc((void**) &dev_values, sizeof(float) * children_cnt);
    hipMemcpy(dev_nodes, nodes, sizeof(node) * children_cnt, hipMemcpyHostToDevice);
    dim3 num_threads(N_CHILDREN, 1, 1);
    alpha_beta_gpu<<<children_cnt, num_threads>>>(dev_nodes, dev_values, depth, AB(-INF, INF));
    float values[children_cnt];
    hipMemcpy(values, dev_values, sizeof(float) * children_cnt, hipMemcpyDeviceToHost);
    int best = std::min_element(values, values + children_cnt) - values;
    hipFree((void**) &dev_values);
    hipFree((void**) &dev_nodes);
    for(int i = 0; i < N_CHILDREN; i++){
        if(is_node[i] && (--children_cnt == 0))
            return i;
    }
    throw "that aint gonna happen";
}


__global__
void compute_children_of_a_node (node *nodes, float *values, node * current_node, unsigned int depth, AB limit)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    //int bl_id = blockIdx.x; //TODO: no blocks so far
    //int th_id = threadIdx.x;
    
    node child;
    node * childptr = &child;
    
    if(get_child(*current_node, id, childptr))
        values[id] = invert( compute_node(child, depth - 1, invert(limit)) );
    else
        values[id] = NODE_INACCESSIBLE;
}

__device__
float compute_node(node const &current_node, unsigned int depth, AB limit)
{
	if(is_terminal(current_node))
		return value(current_node);
	
	node child;
	float best_res = -INF;
	
	for (int i=0; i<N_CHILDREN; i++)
	{
		if(!get_child(current_node, i, &child))
		    continue;
		float temp_res = invert(value(child)); //recursion here should be
		
		if(temp_res > best_res)
		{
			best_res = temp_res;
			if(temp_res > limit.a)
			{
				if(limit.a >= limit.b)
					return INF; 	//alpha-beta prunning - out move is so greate we know B doesn't want
				                    //the parent node. We return INF though in fact best_res should be enough?
	                                //EDIT friday morning. Now I think it should be only best_res, not INF. I'll reconsider it.

	        }
		}
	}
	return best_res;
}
		
    
//we'll paste code from C instead of this function, but will it be any imporvement at all?
__host__
int get_best_index(float * values)
{
	int res_index;
	float val = -1e10;
	for (int i=0; i<N_CHILDREN; i++)
	{
		if(values[i] > val)
		{
			val = values[i];
			res_index = i;
		}
	}
	return res_index;
}

