#include "hip/hip_runtime.h"
#include "alphabeta.h"
#include <algorithm>
#include <iostream>
#include <limits>
#include <cstdio>
    
#define CUDA 1
#define WOJNA 0
#define NIEZABIJANDRZEJA 1

const float INF = std::numeric_limits<float>::infinity();

int dzieci_sewcia, dzieci_krzysia;
int ruch_sewcia;

struct AB {
  float a, b;

  __host__ __device__ AB() {}
  
  __host__ __device__ AB(float a, float b) : a(a), b(b) {}
};

struct stack_entry {

  AB limits;
  node current_node;
  int idx;                                    // id of child to be searched.
  char valid_children[(N_CHILDREN + 7) / 8];  // bit mask of valid children
};

#if CUDA
__host__
#endif
    void
        compute_children_of_a_node(float *values, const node &current_node,
                                   unsigned int depth, AB limit,
                                   dim3 numThreads, int exclude);

#if CUDA
__device__
#endif
    float
        compute_node(node const &current_node, unsigned int depth, AB limit);

__host__ int get_best_index(float *d_values);

__host__ __device__ float alpha_beta_cpu(node const &n, unsigned int depth,
                                         AB limits);

__host__ __device__ float invert(float limit) { return -limit; }
__host__ __device__ AB invert(AB val) {
  return AB(invert(val.b), invert(val.a));
}

/*
 * best_move - can be nullptr if we only want the numerical result
 */
__host__ float alpha_beta(node const &current_node, int depth,
                          unsigned int *best_move_value,
                          dim3 numThreads)  // TODO: for now it's assumed
                                            // N_CHILDREN < legth of "nodes"
                                            // array
{

  if (best_move_value != nullptr) {
#if WOJNA
    ruch_sewcia = false;
#endif
  }

  if (depth == -2 || is_terminal(current_node)) {
    return value(current_node);
  }
  if (depth == -1) {
    return alpha_beta_cpu(current_node, 0, AB(-INF, INF));
  }

  node child;
  float *values = new float[N_CHILDREN];

#if CUDA
  // this variable can probably be deleted
                                        // in the final version, but is crucial
                                        // untill GPU has recursion as well as
                                        // CPU
  int child_cntr = 0;
  struct order_chld {
    float value;
    int idx;
  } ord_nodes[N_CHILDREN];
    
  for (int i = 0; i < N_CHILDREN; i++) 
    if (get_child(current_node, i, &child)) {
      ord_nodes[child_cntr] = {value(child), i};
      ++child_cntr;   
    }
  std::sort(ord_nodes, ord_nodes + child_cntr, 
          [](const order_chld & a, const order_chld & b)->bool
          {
            if(a.value != b.value)
                return a.value > b.value;
            return a.idx > b.idx;
          } );

  get_child(current_node, ord_nodes[0].idx, &child);

  float limit_estimation = invert(alpha_beta(child, depth - 1, nullptr, numThreads));
  compute_children_of_a_node(values, current_node, depth + 1,
                             AB(limit_estimation, INF), numThreads, ord_nodes[0].idx);
#else
  compute_children_of_a_node(d_nodes, values, &current_node, depth,
                             AB(-INF, INF), 0);
  
#endif

 // int best_ind = get_best_index(velues);
  int best_ind = std::max_element(values, values + N_CHILDREN) - values;

  // This may happen if all the branches were prunned, we must than send
  // any move so that is it prunned in the parent node.
  // We only need to find a valid one
  if (!get_child(current_node, best_ind, nullptr)) {
    for (int i = 0; i < N_CHILDREN; i++)
      if (get_child(current_node, i, &child)) {
        best_ind = i;
        break;
      }
  }

  float result = values[best_ind];

#if CUDA
  if (result <= limit_estimation) best_ind = ord_nodes[0].idx;
#endif

  if (best_move_value != nullptr) *best_move_value = best_ind;
  delete[] values;

  if (best_move_value != nullptr) {
#if WOJNA
    std::cout << "Krzys visited " << dzieci_krzysia << " nodes\n";
#endif
  }
  return result;
}

const int MAX_STACK_SIZE = 10;

#define DEBUG if (0)
__global__ void alpha_beta_gpu(node *nodes, float *values, unsigned int depth,
                               AB limits_) {

  __shared__ stack_entry stack[MAX_STACK_SIZE];
  __shared__ stack_entry *stacklast;
  __shared__ unsigned int valid_children[N_CHILDREN];
  __shared__ bool toContinue;
  __shared__ float children_values[N_CHILDREN];

  int thid = threadIdx.x;
  int blid = blockIdx.x;
  node local_node;
  float ret;

  valid_children[thid] = 0;

  if (thid == 0) {

    stack[0].limits = limits_;
    stack[0].current_node = nodes[blid];
    stack[0].idx = 0;

    stacklast = stack;
    DEBUG printf("%d alfabeta(%f,%f,(%lx,%lx)) initial call\n", blid,
                 stacklast->limits.b, stacklast->limits.a,
                 stacklast->current_node.os, stacklast->current_node.xs);
  }
  __syncthreads();
  while (stacklast >= stack) {
    if (thid == 0) {
      toContinue = false;
      if (stacklast->current_node.os & stacklast->current_node.xs) {
        DEBUG printf("%d This is wrong %lx %lx\n", blid,
                     stacklast->current_node.os, stacklast->current_node.xs);
      }
    }

    if (is_terminal(stacklast->current_node)) {  // if current node is terminal
      if (thid == 0) {
        DEBUG printf("%d node is terminal\n", blid);
        float val = value(stacklast->current_node);
        DEBUG printf("%d val = %f\n", blid, val);
        ret = val;
        stacklast--;
        toContinue = true;
      }
    } else if (stacklast == stack + depth) {  // if max depth reached
      DEBUG printf("%d max depth reached.\n", blid);

      if (get_child(stacklast->current_node, thid,
                    &local_node)) {  // find values of children
        children_values[thid] = value(local_node);
      } else {
        children_values[thid] = INF;
      }
#if WOJNA
      if (ruch_sewcia)
        dzieci_sewcia++;
      else
        dzieci_krzysia++;
#endif

      DEBUG {
        printf("%d children values: ", blid);
        for (int i = 0; i < N_CHILDREN; i++)
          printf("%d[%d]:%.0f ", blid, i, children_values[i]);
        printf("\n");
      }
      __syncthreads();

      for (int d = 1; d < N_CHILDREN; d <<= 1) {

        if ((thid ^ d) < N_CHILDREN) {  // find min  of these values
          __syncthreads();
          float vald = children_values[thid ^ d];
          float val = children_values[thid];
          __syncthreads();
          if (vald < val) children_values[thid] = vald;
        }
      }

      if (thid == 0) {
        ret = -children_values[0];
        stacklast--;
        DEBUG printf("%d min = %f (return)\n", blid, ret);
        toContinue = true;
      }
    }

    __syncthreads();

    if (toContinue) {
      continue;
    }

    if (stacklast->idx ==
        0) {  // if this is the first time we are at current node
      DEBUG printf("%d first time in the node\n", blid);
      bool has_child =
          get_child(stacklast->current_node, thid, nullptr) ? (thid & 0xf) : 0;
      // we want to calculate bit mask of valid children. First we will find
      // singleton masks
      // and then we will run (bitwise or)-scan.
      valid_children[thid] = has_child ? (1 << (thid & 7)) : 0;

      // if(blid == 0) printf("%d %d vs[%d]=%x\n", counter++, blid, thid,
      // valid_children[thid]);
      __syncthreads();

      // note that no syncthreading is needed below - communication is within
      // warp;
      for (int d = 1; d < 8; d <<= 1) {
        if ((thid ^ d) < N_CHILDREN) {
          valid_children[thid] =
              valid_children[thid] | valid_children[thid ^ d];
        }
      }

      __syncthreads();

      if ((thid & 7) == 0) {
        stacklast->valid_children[thid >> 3] = valid_children[thid];
      }

      __syncthreads();

      DEBUG {
        for (int i = 0; i < (N_CHILDREN + 7) / 8; i++)
          printf("%x", 0xff & stacklast->valid_children[i]);
        printf("\n");
      }
      __syncthreads();
    } else {  // we've just returned from recursion.

      DEBUG printf("%d just returned from recursion\n", blid);
      if (thid == 0 && -ret > stacklast->limits.a) {
        stacklast->limits.a = -ret;
        DEBUG printf("%d now alpha = %f\n", blid, stacklast->limits.a);
      }
      if (stacklast->limits.a >= stacklast->limits.b) {  // pruning
        stacklast->idx = N_CHILDREN;
      }
    }

    __syncthreads();

    if (thid == 0) {
      int idx = stacklast->idx;
      // find valid idx
      while (idx < N_CHILDREN &&
             ((stacklast->valid_children[idx >> 3] >> (idx & 7)) & 1) == 0) {

        idx++;
      }

      if (idx ==
          N_CHILDREN) {  // if all children searched - return from recursion
        DEBUG printf("%d return %f\n", blid, stacklast->limits.a);
        ret = stacklast->limits.a;
        stacklast--;
      } else {  // otherwise search children.

        (stacklast + 1)->limits.a = -stacklast->limits.b;
        (stacklast + 1)->limits.b = -stacklast->limits.a;
        get_child(stacklast->current_node, idx, &(stacklast + 1)->current_node);
        (stacklast + 1)->idx = 0;

        stacklast->idx = ++idx;
        stacklast++;
        DEBUG printf("%d alfabeta(%f,%f,(%lx,%lx))\n", blid,
                     stacklast->limits.b, stacklast->limits.a,
                     stacklast->current_node.os, stacklast->current_node.xs);
      }
    }
    __syncthreads();
  }
  if (thid == 0) values[blid] = ret;
}

__host__ __device__ float alpha_beta_cpu(node const &n, unsigned int depth,
                                         AB limits) {
  node c;
  if (is_terminal(n)) return value(n);
  if (depth == 0) {
    float min_val = INF;
    for (int i = 0; i < N_CHILDREN; i++) {
      if (get_child(n, i, &c)) {
        float val = value(c);
        if (val < min_val) min_val = val;
      }
    }
    return -min_val;
  }
  float best_val = -INF;
  for (int i = 0; i < N_CHILDREN; i++) {
    if (get_child(n, i, &c)) {
      float val = -alpha_beta_cpu(c, depth - 1, AB(-limits.b, -limits.a));
      if (val > best_val) best_val = val;
      if (val > limits.a) limits.a = val;
      if (limits.a >= limits.b) break;
    }
  }
  return best_val;
}

extern const int DEPTH;
unsigned int get_alpha_beta_cpu_kk_move(const node &n) {
  unsigned int res;
  dim3 dm3_unused;
  alpha_beta(n, DEPTH, &res, dm3_unused);
  return res;
}

unsigned int get_alpha_beta_gpu_move(node const &n) {
#if WOJNA
  ruch_sewcia = true;
#endif

  const int depth = DEPTH;
  unsigned int moves[N_CHILDREN];
  node nodes[N_CHILDREN];
  int children_cnt = 0;

  for (unsigned int i = 0; i < N_CHILDREN; i++) {
    if (get_child(n, i, &nodes[children_cnt])) moves[children_cnt++] = i;
  }

  node *dev_nodes;
  float *dev_values;
  hipMalloc((void **)&dev_nodes, sizeof(node) * children_cnt);
  hipMalloc((void **)&dev_values, sizeof(float) * children_cnt);
  hipMemcpy(dev_nodes, nodes, sizeof(node) * children_cnt,
             hipMemcpyHostToDevice);
  dim3 num_threads(N_CHILDREN, 1, 1);
  alpha_beta_gpu << <children_cnt, num_threads>>>
      (dev_nodes, dev_values, depth, AB(-INF, INF));
  float values[children_cnt];
  hipMemcpy(values, dev_values, sizeof(float) * children_cnt,
             hipMemcpyDeviceToHost);
  hipFree((void **)&dev_values);
  hipFree((void **)&dev_nodes);
  int best = std::min_element(values, values + children_cnt) - values;
#if WOJNA
  std::cout << "Sewcio visited " << dzieci_sewcia << " nodes\n";
#endif
  return moves[best];
}
unsigned int get_alpha_beta_cpu_move(node const &n) {
  const int depth = DEPTH;
  unsigned int moves[N_CHILDREN];
  node nodes[N_CHILDREN];
  int children_cnt = 0;

  for (unsigned int i = 0; i < N_CHILDREN; i++) {
    if (get_child(n, i, &nodes[children_cnt])) moves[children_cnt++] = i;
  }
  float values[children_cnt];
  AB ab(-INF, INF);
  for (int i = 0; i < children_cnt; i++) {
    values[i] = alpha_beta_cpu(nodes[i], depth, invert(ab));
  }
  int best = std::min_element(values, values + children_cnt) - values;

  return moves[best];
}

#if CUDA
__host__
#endif
    void
compute_children_of_a_node(float *values, const node &current_node,
                           unsigned int depth, AB limit, dim3 numThreads, int exclude) {
#if !CUDA
  node child;
  node *childptr = &child;
  for (int id = 0; id < N_CHILDREN; id++)
    if (get_child(current_node, id, childptr))
      values[id] = invert(compute_node(child, depth - 1, invert(limit)));
    else
      values[id] = -INF;

#else

  node *nodes = new node[N_CHILDREN];
  for (int i = 0; i < N_CHILDREN; i++) values[i] = -INF;

  unsigned int moves[N_CHILDREN];
  int children_cnt = 0;

  for (unsigned int i = 0; i < N_CHILDREN; i++) {
    if (get_child(current_node, i, &nodes[children_cnt]) && i != exclude)
      moves[children_cnt++] = i;
  }

  node *d_nodes;
  float *d_values;
  hipMalloc((void **)&d_nodes, sizeof(node) * children_cnt);
  hipMalloc((void **)&d_values, sizeof(float) * children_cnt);
  hipError_t cudaResult = hipMemcpy(
      d_nodes, nodes, sizeof(node) * children_cnt, hipMemcpyHostToDevice);
  if (cudaResult != hipSuccess) {
    printf("cuda memcpy error %s\n", hipGetErrorString(cudaResult));
    throw 1;
  }

  alpha_beta_gpu << <children_cnt, numThreads>>>
      (d_nodes, d_values, depth - 1, invert(limit));

  cudaResult = hipDeviceSynchronize();
  if (cudaResult != hipSuccess) {
    printf("cuda synchronize error %s\n", hipGetErrorString(cudaResult));
    throw 1;
  }

  cudaResult = hipMemcpy(values, d_values, sizeof(float) * children_cnt,
                          hipMemcpyDeviceToHost);
  if (cudaResult != hipSuccess) {
    printf("cuda memcpy error %s\n", hipGetErrorString(cudaResult));
    throw 1;
  }

  for (int i = children_cnt - 1; i >= 0; i--) {
    values[moves[i]] = -values[i];
    values[i] = -INF;
  }
  hipFree(d_nodes);
  hipFree(d_values);

#endif
}

#if CUDA
__device__
#endif
    float
compute_node(node const &current_node, unsigned int depth, AB limit) {
  if (depth == 0 || is_terminal(current_node)) return value(current_node);

  node child;
  float best_res = INF;

  for (int i = 0; i < N_CHILDREN; i++) {
    if (!get_child(current_node, i, &child)) continue;
#if CUDA
    float temp_res = invert(value(child));
#else
    float temp_res = invert(compute_node(child, depth - 1, invert(limit)));
#endif

    if (temp_res > best_res) {
      best_res = temp_res;
      if (temp_res > limit.a) {
        limit.a = temp_res;
        if (limit.a >= limit.b) return best_res;
      }
    }
  }
  return best_res;
}

__host__ int get_best_index(float *values) {
  int res_index = -1;
  float val = -1e10;
  for (int i = 0; i < N_CHILDREN; i++) {
    if (values[i] > val) {
      val = values[i];
      res_index = i;
    }
  }
  return res_index;
}
