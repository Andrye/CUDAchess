#include "hip/hip_runtime.h"

#include "alphabeta.h"
#include "node.h"
#include <cstdint>
#include <iostream>
#include <vector>

const int DEPTH = 5;

unsigned int get_bots_move(node const&);


node launchKernel(node const& current_node){
    const int n_threads = 1024;

    node* nodes = new node[n_threads];
    for(int i = 0; i < n_threads; i++){
	    nodes[i] = {};
    }

    node *dev_nodes;
    float* dev_values;

    hipMalloc((void**) &dev_values, sizeof(float) * n_threads);
    hipMalloc((void**) &dev_nodes, sizeof(node) * n_threads); //TODO: so far I decided that it indeed should be n_threads, not n_blocks. We'll see later.
    hipMemcpy(dev_nodes, nodes, sizeof(node) * n_threads, hipMemcpyHostToDevice);
    dim3 numThreads(n_threads, n_children, 1);

    node best_move;
    alpha_beta(dev_nodes, dev_values, current_node, DEPTH, n_children, &best_move, numThreads); //TODO: for now it's hipDeviceSynchronize();
    delete[] nodes;
    return best_move;
}

int main(){
    node nodes[2];
    nodes[0] = {};
    for(int i = 0; !is_terminal(nodes[i]); i=1-i){
        unsigned int move;
	    if(i==0)
	        move = get_console_move(nodes[i]);
	    else
	        move = get_bots_move(nodes[i]);
	    if(!get_child(nodes[i], move, nodes+1-i))
	        throw "Wrong move returned";
    }
    printf("Implement me\n");
    return 0;
}



unsigned int get_bots_move(node const &n)
{
    node next_node = launchKernel(n);
    for(unsigned int i = 0; i < n_children; i++)
        if(get_child(n, i, nullptr))
            return i;
    throw "no move can be done";
}
