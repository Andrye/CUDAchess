#include "hip/hip_runtime.h"

#include "alphabeta.h"
#include "node.h"
#include <cstdint>
#include <iostream>
#include <map>
#include <vector>
#include <chrono>

extern const int DEPTH = 2;

unsigned int get_bots_move(node const&);

unsigned int launchKernel(node const& current_node){
    const int n_threads = N_CHILDREN;


 /*   node *dev_nodes;
    float* dev_values;

    hipMalloc((void**) &dev_values, sizeof(float) * n_threads); //after we incorporate scan, this array should be moved to shared memeory
    hipMalloc((void**) &dev_nodes, sizeof(node) * n_threads); //unused. According to Sewcio should be * n_blocks not n_children
    hipMemcpy(dev_nodes, nodes, sizeof(node) * n_threads, hipMemcpyHostToDevice);
   */
    dim3 numThreads(n_threads, 1/*n_children*/, 1);

    unsigned int best_move;
 //   alpha_beta(dev_nodes, dev_values, current_node, DEPTH, &best_move, numThreads); //TODO: for now it's hipDeviceSynchronize();
    alpha_beta(current_node, DEPTH, &best_move, numThreads);
    
    /*delete[] nodes;
    hipFree((void**) &dev_values);
    hipFree((void**) &dev_nodes); //TODO: so far I decided that it indeed should be n_threads, not n_blocks. We'll see later.
    */
    return best_move;
}


typedef unsigned int (*strategy)(node const&);
typedef std::chrono::duration<double> time_interv;

unsigned int count_time(strategy player, node const& n, time_interv *player_time)
{
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();
    
    unsigned int move = player(n);

    end = std::chrono::system_clock::now();
    *player_time += end-start;
    
    return move;
    //std::time_t end_time = std::chrono::system_clock::to_time_t(end);
}

int main(int argc, char *argv[]){
    std::map<std::string, strategy> players = {
        {"stdin", get_console_move},
        {"cpu", get_alpha_beta_cpu_move},
        {"gpu", get_alpha_beta_gpu_move},
        {"gpukk", launchKernel}
    };
    if(argc < 3 || !players.count(argv[1]) || !players.count(argv[2])){
        std::cout << "Usage: " << argv[0] << " player1 player2 [ascetic_display = 1 [depth = 4 ]]" << std::endl;
        std::cout << "\twhere\n- player1, player 2 is one of:";
        for(auto p : players)
            std::cout << " " << p.first;
        std::cout << "\n- full_display set to 0 means printing only the time used by the players";
        std::cout << "\n- depth >= 2";
        std::cout << std::endl;
        return 0;
    }


    if(argc >= 4)
    {
        full_display(argv[3]);
        if(argc >= 5)
        {

    auto player1 = players[argv[1]];
    auto player2 = players[argv[2]];
    node nodes[2];
    nodes[0] = {};
    int i;
    
    time_interv pl1_time(0), pl2_time(0);

    for(i = 0; !is_terminal(nodes[i]); i=1-i){
        unsigned int move;
	    std::cout << nodes[i] << "Node value: " << value(nodes[i]) << std::endl;
	    if(i==0)
	        move = count_time(player1, nodes[i], &pl1_time);
	    else
	        move = count_time(player2, nodes[i], &pl2_time);
	    if(!get_child(nodes[i], move, nodes+1-i))
        {
            printf("move wrong %d\n", move);
            throw "Wrong move returned";
        }
    }
    std::cout << "GAME OVER. Player " << (i==1 ? "1 (O)" : "2 (X)") << " won!" << std::endl << nodes[i];
    std::cout << "Player 1 (" << argv[1] << ") took " << pl1_time.count() << "\n";
    std::cout << "Player 2 (" << argv[2] << ") took " << pl2_time.count() << "\n";
    return 0;
}



unsigned int get_bots_move(node const &n)
{
    return launchKernel(n);
    /*for(unsigned int i = 0; i < n_children; i++)
        if(get_child(n, i, nullptr))
            return i;
    throw "no move can be done";
*/
}
